#include <hip/hip_runtime.h>
#include <chrono>

#include <iostream>

#define N 2000000000  // 向量大小

// CUDA 核心函数：向量加法
__global__ void vectorAdd(int8_t *A, int8_t *B, int8_t *C, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        C[index] = A[index] + B[index];
    }
}

int main() {
    int8_t *h_A, *h_B, *h_C;  // 主机向量
    int8_t *d_A, *d_B, *d_C;  // 设备向量

    size_t bytes = N * sizeof(int8_t);

    // 分配主机内存
    hipHostAlloc((void**)&h_A, bytes, hipHostMallocMapped);
    hipHostAlloc((void**)&h_B, bytes, hipHostMallocMapped);
    hipHostAlloc((void**)&h_C, bytes, hipHostMallocMapped);

    // 初始化主机向量
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<int8_t>(1);
        h_B[i] = static_cast<int8_t>(1);
    }

    // 分配设备内存
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // 复制主机向量到设备
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    //warmup
    int threads = 1024;
    int runtimes = 500;
    int blocks = (N + threads - 1) / threads;
    vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, N);
    std::cout << "start" << std::endl;
    struct timespec startTimeSpec;
    clock_gettime(CLOCK_REALTIME, &startTimeSpec);

    // 启动 CUDA 核心
    for(int i=0;i<runtimes;i++){
      auto start = std::chrono::system_clock::now();
      vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, N);
      auto end = std::chrono::system_clock::now();
    //   std::cout << end - start << " ms" << std::endl;
    }

    struct timespec stopTimeSpec;
    clock_gettime(CLOCK_REALTIME, &stopTimeSpec);

    // 计算耗时
    //double clientTimeDelta = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()/1000.0;
        double clientTimeDelta =
                        (double)stopTimeSpec.tv_sec +
                        (double)stopTimeSpec.tv_nsec / 1000000000.0 -
                        ((double)startTimeSpec.tv_sec +
                         (double)startTimeSpec.tv_nsec / 1000000000.0);
    // 计算总操作数（两个向量加法）
    double totalOperations = N*runtimes; // 每个加法操作算一次
    // 计算 TOPS
    float tops = (double)((unsigned long long int)runtimes *
                                 N) /
                        clientTimeDelta / 1000.0 / 1000.0 / 1000.0 / 1000.0 / 2; // 转换为 Tera Operations
    std::cout << "计算性能: " << tops << " TOPS" << std::endl;
    std::cout << "耗时:" << clientTimeDelta*1000 << std::endl;

    // 复制结果到主机
    return;
    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    bool correct = true;
    for (int i = 0; i < 100; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            correct = false;
            break;
        }
    }

    if (correct) {
        std::cout << "向量加法完成，结果正确！" << std::endl;
    } else {
        std::cout << "结果错误！" << std::endl;
    }

    // 清理内存
    //cudaFree(d_A);
    //cudaFree(d_B);
    //cudaFree(d_C);
    //free(h_A);
    //free(h_B);
    //free(h_C);

    return 0;
}

